#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void matrixMul(const double* A, const double* B, double* C, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n)
    {
        double sum = 0.0;
        for (int k = 0; k < n; k++)
            sum += A[row * n + k] * B[k * n + col];
        C[row * n + col] = sum;
    }
}

int main(int argc, char* argv[])
{
    if (argc != 2)
    {
        fprintf(stderr, "Uso: %s <dimensione matrice quadrata>\n", argv[0]);
        return EXIT_FAILURE;
    }

    int N = atoi(argv[1]);
    if (N <= 0)
    {
        fprintf(stderr, "Errore: la dimensione deve essere un intero positivo\n");
        return EXIT_FAILURE;
    }

    size_t size = N * N * sizeof(double);

    double *h_A = (double*)malloc(size);
    double *h_B = (double*)malloc(size);
    double *h_C = (double*)malloc(size);

    if (!h_A || !h_B || !h_C)
    {
        fprintf(stderr, "Errore allocazione host memory\n");
        return EXIT_FAILURE;
    }

    for (int i = 0; i < N * N; i++)
    {
        h_A[i] = 1.0;
        h_B[i] = 2.0;
    }

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (N + 15) / 16);

    matrixMul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("C[0][0] = %f\n", h_C[0]);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;
}
