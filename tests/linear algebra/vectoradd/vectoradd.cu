
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define CUDA_SAFECALL(call)                                             \
{                                                                       \
    call;                                                               \
    hipError_t err = hipGetLastError();                                 \
    if (hipSuccess != err) {                                           \
        fprintf(                                                        \
            stderr,                                                     \
            "Cuda error in function '%s' file '%s' in line %i : %s.\n", \
            #call, __FILE__, __LINE__, hipGetErrorString(err));        \
        fflush(stderr);                                                 \
        exit(EXIT_FAILURE);                                             \
    }                                                                   \
}


__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) c[id] = a[id] + b[id];
}

int main(int argc, char *argv[])
{
    int n = 100000;
    if (argc > 1) n = atoi(argv[1]);

    double *h_a;
    double *h_b;
    double *h_c;

    double *d_a;
    double *d_b;
    double *d_c;

    size_t bytes = n * sizeof(double);

    h_a = (double *)malloc(bytes);
    h_b = h_a;
    h_c = h_a;

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    for (int i = 0; i < n; i++)
        h_a[i] = sin(i) * sin(i);
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);

    for (int i = 0; i < n; i++)
        h_b[i] = cos(i) * cos(i);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    for (int i = 0; i < n; i++)
        h_c[i] = 0;
    hipMemcpy(d_c, h_c, bytes, hipMemcpyHostToDevice);   

    int blockSize = 1024;
    int gridSize = (int)ceil((float)n / blockSize);

    CUDA_SAFECALL((vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n)));

    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    double sum = 0;
    for (int i = 0; i < n; i++) sum += h_c[i];
    printf("Final sum = %f; sum/n = %f (should be ~1)\n", sum, sum / n);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);

    return 0;
}
